
#include <hip/hip_runtime.h>
#include <cstdint>
#include <math.h>

#define BLOCK_SIZE 16

template<typename T>
__device__ T* get_2d_array_element(T* _2d_array_base, uint32_t row, uint32_t column, size_t pitch)
{
    T* p_result = (T*)((char*)_2d_array_base + row*pitch) + column;
    return p_result;
}


__device__ __constant__ float GX_kernel[3][3] = { {-1, 0, 1}, {-2, 0, 2}, {-1, 0, 1} };
__device__ __constant__ float GY_kernel[3][3] = { {-1, -2, -1}, {0, 0, 0}, {1, 2, 1} };

extern "C"
{
    __global__ void sobel_filter(const float* __restrict__ img, float* __restrict__ sobel,
        size_t order_x, size_t order_y, size_t kernel_size, size_t bytes_per_float)
 {

        int row = BLOCK_SIZE * blockIdx.x + threadIdx.x;
        int column = BLOCK_SIZE * blockIdx.y + threadIdx.y;

        int pitch = order_y * bytes_per_float;

        __shared__ float I[BLOCK_SIZE][BLOCK_SIZE];

        if ((row >= order_x) || (column >= order_y) || (row < 0) || (column < 0))
        {
            I[threadIdx.x][threadIdx.y] = 0.0;
            return;
        }

        int xi = 0;
        int yi = 0;

        float sx = 0;
        float sy = 0;

        I[threadIdx.x][threadIdx.y] = *get_2d_array_element(img, row, column, pitch);
        __syncthreads();

        for (xi=-1; xi<2; xi++)
        {
            for (yi=-1; yi<2; yi++)
            {
                if ((row+xi >= order_x) || (column+yi >= order_y) || (row+xi < 0) || (column+yi < 0))
                {
                    sx+=0.0;
                    sy+=0.0;
                }
                else
                {
            if ((threadIdx.x + xi < 0) || (threadIdx.x + xi > BLOCK_SIZE-1) || (threadIdx.y + yi > BLOCK_SIZE-1) || (threadIdx.y + yi < 0))
                  {
                    sx+= *get_2d_array_element(img, row+xi, column+yi, pitch) * GX_kernel[xi+1][yi+1];
                    sy+= *get_2d_array_element(img, row+xi, column+yi, pitch) * GY_kernel[xi+1][yi+1];
                  }
                    else
                    {
                      sx+= I[threadIdx.x+xi][threadIdx.y+yi] * GX_kernel[xi+1][yi+1];
                      sy+= I[threadIdx.x+xi][threadIdx.y+yi] * GY_kernel[xi+1][yi+1];
                    }
                }
            }
        }

        *get_2d_array_element(sobel, row, column, pitch) = sqrt(sx*sx + sy*sy);
 }

}
